#include "util.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

void check_cuda(const char* file, int line) {
  hipError_t code = hipPeekAtLastError();
  if (code != 0) {
    char buf[1024];
    sprintf(buf, "Cuda error: %s\n", hipGetErrorString(code));
    throw VMException(buf, file, line);
  }
}
