#include "bytecode.h"

Program::~Program() {
  if (_gpu_ptr) {
    hipFree(_gpu_ptr);
  }
}

void* Program::to_gpu() {
  if (_gpu_ptr) {
    return _gpu_ptr;
  }
  hipMalloc(&_gpu_ptr, nbytes());
  hipMemcpy(_gpu_ptr, &_ops[0], nbytes(), hipMemcpyHostToDevice);
  return _gpu_ptr;
}

