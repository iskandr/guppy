#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <vector>

/*
#define BYTECODE_OP static inline __device__

BYTECODE_OP void load_slice() {

}

BYTECODE_OP void add(void* a, void *b) {

}
*/
double Now() {
    timespec tp;
    clock_gettime(CLOCK_MONOTONIC, &tp);
    return tp.tv_sec + 1e-9 * tp.tv_nsec;
}

#define TIMEOP(op)\
{\
  double st = Now();\
  op;\
  double ed = Now();\
  fprintf(stderr, "%s finished in %.f seconds.\n", #op, end - start);\
}


enum OP_CODE {
  LOAD_SLICE, STORE_SLICE,    // load slice of global arrays into shared vector
  LOAD_SCALAR, STORE_SCALAR, // distribute scalar across elements of shared vector
  ADD, SUB, MUL, DIV,        // arithmetic between shared vectors
  BAD
};

struct Op {
  Op() : code(BAD), x(0), y(0), z(0) {}
  Op(OP_CODE code, int x, int y, int z) : code(code), x(x), y(y), z(z)  {}

  OP_CODE code; 
  int x, y, z;
};

struct Program {
	std::vector<Op> _ops;
	Op* _gpu_ptr;


	Program& Add(int x, int y, int z) {
		_ops.push_back(Op(ADD, x, y, z));
		return *this;
	}
	Program& LoadSlice(int src, int dst) {
		_ops.push_back(Op(LOAD_SLICE, src, dst, 0));
		return *this;
	}
	Program& StoreSlice(int src, int dst) {
		_ops.push_back(Op(STORE_SLICE, src, dst, 0));
		return *this;
	}

	int size() {
      return _ops.size();
	}

	int nbytes () {
	  return sizeof(Op) * this->size();
	}

	Op* host_ptr() {
	  return &_ops[0];
	}
	Op* to_gpu() {
	  if (_gpu_ptr) {
		  return _gpu_ptr;
	  }
	  hipMalloc(&_gpu_ptr, this->nbytes());
	  hipMemcpy(_gpu_ptr, this->host_ptr(), this->nbytes(), hipMemcpyHostToDevice);
	  return _gpu_ptr;
	}

	Program() : _gpu_ptr(NULL) {}
	~Program () {
	  if (_gpu_ptr) {
	    hipFree(_gpu_ptr);
	  }
	}
};


struct Vec {
  int _n;
  int _nbytes;
  float* _host_data;
  float* _gpu_data;
  bool _host_dirty;
  bool _gpu_dirty;

  void init(int n) {
    _n = n;
    _nbytes = sizeof(float) * n
    		;
     hipHostMalloc(&_host_data, this->_nbytes, hipHostMallocDefault);
     hipMalloc(&_gpu_data, this->_nbytes);
    _host_dirty = false;
    _gpu_dirty = true;
  }

  Vec(int n) {
    this->init(n);
  }

  Vec (int n, float fill_value) {
    this->init(n);
    for (int i = 0; i < n; ++i) {
      _host_data[i] = fill_value;
    }
  }

  float* get_gpu_data() {
     if (_gpu_dirty) { this->copy_to_gpu(); }
     _host_dirty = true;
     _gpu_dirty = false;
     return _gpu_data;
  }

  float* get_host_data() {
	  if (_host_dirty) { this->copy_to_host();}
	  _gpu_dirty = true;
	  _host_dirty = false;
	  return _host_data;
  }

  void copy_to_host() {
	  hipMemcpy(this->_host_data, this->_gpu_data, this->_nbytes, hipMemcpyDeviceToHost);
  }


  void copy_to_gpu() {
	  hipMemcpy(this->_gpu_data, this->_host_data, this->_nbytes, hipMemcpyHostToDevice);
  }

  ~Vec() {
	  hipFree(_gpu_data);
	  hipHostFree(_host_data);
  }

};

// NOT YET USING 2D blocks
#define THREADS_X 512
#define THREADS_Y 1


#define THREADS_PER_BLOCK (THREADS_X * THREADS_Y)
#define REGISTER_WIDTH THREADS_PER_BLOCK
#define NUM_REGISTERS 4



__global__ void run(
		Op* program, long n_ops,
		float** values, long n_args,
		float* constants, long n_consts) {

  __shared__ float registers[NUM_REGISTERS][REGISTER_WIDTH];

  int block_offset = blockIdx.x * blockDim.x;
  int local_idx = threadIdx.x;
int global_idx = block_offset + local_idx;

  for (int pc = 0; pc < n_ops; ++pc) {
    Op op = program[pc];
    switch (op.code) {
    case LOAD_SLICE: {
      registers[op.y][local_idx] = values[op.x][global_idx];
    }
    break;

    case STORE_SLICE: {
      values[op.y][global_idx] = registers[op.x][local_idx];
    }
    break;

	case ADD: {
	    float x = registers[op.x][local_idx]; //+ startIdx + threadIdx.x;
	    float y = registers[op.y][local_idx]; //+ startIdx + threadIdx.x;
	    registers[op.z][local_idx] = x + y;
      }
	break;
    }  
  }
}



int main(int argc, const char** argv) { 
  int N = 400 * THREADS_PER_BLOCK;
  if (argc > 1) {
    N = strtol(argv[1], NULL, 10);
  }
    
  Vec a(N, 1.0);
  Vec b(N, 2.0);
  Vec c(N);
  
  const int n_values = 3;
  float* h_values[n_values];
  h_values[0]= a.get_gpu_data();
  h_values[1] = b.get_gpu_data();
  h_values[2] = c.get_gpu_data();

  float** d_values;
  hipMalloc(&d_values, sizeof(float*) * n_values);
  hipMemcpy(d_values, h_values, sizeof(float*) * n_values, hipMemcpyHostToDevice);


  Program h_program;

  h_program.
    LoadSlice(0,0).
    LoadSlice(1,1).
    Add(0,1,2).
    StoreSlice(2,2);

  double st = Now();
  run<<<N / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(
		  h_program.to_gpu(), h_program.size(),
		  d_values, n_values,
		  0, 0);
  hipDeviceSynchronize();
  double ed = Now();
  fprintf(stderr, "%.5f seconds\n", ed -st);

  float* ad = a.get_host_data();
  printf("%f %f %f\n", ad[0], ad[10], ad[200]);
  float* bd = b.get_host_data();
  printf("%f %f %f\n", bd[0], bd[10], bd[200]);
  float* cd = c.get_host_data();
  printf("%f %f %f\n", cd[0], cd[10], cd[200]);
  return 0; 
}
