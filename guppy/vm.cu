#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <vector>
#include <math.h>


#include "bytecode.h"
#include "vec.h"
#include "util.h"

#undef _GLIBCXX_ATOMIC_BUILTINS
#undef _GLIBCXX_USE_INT128
static const int kThreadsX = 8; // 16;
static const int kThreadsY = 4; // 16;

// seems to give slightly better performance than kOpsPerThread = 8
static const int kOpsPerThread = 9;

static const int kThreadsPerBlock = kThreadsX * kThreadsY;

static const int kRegisterWidth = kThreadsPerBlock * kOpsPerThread;

static const int kNumVecRegisters = 4;

static const int kNumIntRegisters = 10;
static const int kNumFloatRegisters = 10;

static const int kMaxProgramLength = 1000; 

#define PREFETCH_GPU_BYTECODE 

__global__ void run(char* program,
                    long program_nbytes,
                    float** values,
                    long n_args,
                    float* constants,
                    long n_consts) {

  
  // making vector slightly longer seems to minorly improve 
  // performance -- due to bank conflicts? 
  __shared__ float vectors[kNumVecRegisters][kRegisterWidth+1];

  __shared__ int   int_scalars[kNumIntRegisters];
  __shared__ float float_scalars[kNumFloatRegisters];
  

  const int block_offset = blockIdx.y * gridDim.x + blockIdx.x;
  const int local_idx = threadIdx.y * blockDim.x + threadIdx.x;
  const int block_start_idx = block_offset * kRegisterWidth;
  const int global_idx = block_start_idx + (local_idx * kOpsPerThread);

 
  #ifdef PREFETCH_GPU_BYTECODE 
    /* preload program so that we don't make 
       repeated global memory requests 
    */  
    __shared__ char  cached_program[kMaxProgramLength];
    for (int i = local_idx; i < program_nbytes; i+=kThreadsPerBlock) {
      cached_program[i] = program[i];      
    }  
  #endif 
  // by convention, the first int register contains the global index
  int_scalars[BlockStart] = block_offset; 
  int_scalars[VecWidth] = kRegisterWidth;
  int_scalars[BlockEltStart] = block_offset * kRegisterWidth; 

  int pc = 0;
  Instruction* instr;
  while (pc < program_nbytes) {
    
    #ifdef PREFETCH_GPU_BYTECODE 
      instr = (Instruction*) &cached_program[pc];
    #else
      instr = (Instruction*) &program[pc]; 
    #endif 
    pc += instr->size;

    switch (instr->code) {
    case LoadVector::op_code: {
      LoadVector* load_slice = (LoadVector*) instr;
      
      float* reg = vectors[load_slice->target_vector]; 
      const float* src = values[load_slice->source_array];
      const int start = int_scalars[load_slice->start_idx] + local_idx;
      int nelts = int_scalars[load_slice->nelts];
      nelts = nelts <= kRegisterWidth ? nelts : kRegisterWidth; 
      for (int i = 0; i < nelts; i += kThreadsPerBlock) { 
        const float elt = src[start+i]; 
        reg[i] = elt;
      }
      break;
    }

    case StoreVector::op_code: {
      StoreVector* store_vector = (StoreVector*) instr;
      const float* reg = vectors[store_vector->source_vector];
      float* dst = values[store_vector->target_array];
      const int start = int_scalars[store_vector->start_idx] + local_idx;
      int nelts = int_scalars[store_vector->nelts];
      nelts = nelts <= kRegisterWidth ? nelts : kRegisterWidth; 
      for (int i = 0; i < nelts; i += kThreadsPerBlock) { 
        const float elt = reg[i]; 
        dst[i+start] = elt; 
      }
      break;
    }

    case Add::op_code: {
      Add* add = (Add*) instr;
      const float* a = vectors[add->arg1];
      const float* b = vectors[add->arg2];
      float *c = vectors[add->result];
      for (int i = local_idx; i < kRegisterWidth; i += kOpsPerThread) {
        c[i] = a[i] + b[i];
      }
      break;
    }
    
    case Map::op_code: {
      Map* map = (Map*) instr;
      /*
      const float* reg = registers[op.x]; 
      for (int i = local_idx; i < kRegisterWidth; i += kOpsPerThread) {
        elt = reg[i];

      }
      */
      break;
    }
    }
  }
}

int main(int argc, const char** argv) {
  int N = 10000 * kRegisterWidth; //2 << 24;

  Vec a(N, 1.0);
  Vec b(N, 2.0);
  Vec c(N);

  const int n_values = 3;
  float* h_values[n_values];
  h_values[0] = a.get_gpu_data();
  h_values[1] = b.get_gpu_data();
  h_values[2] = c.get_gpu_data();

  float** d_values;
  hipMalloc(&d_values, sizeof(float*) * n_values);
  hipMemcpy(d_values, h_values, sizeof(float*) * n_values, hipMemcpyHostToDevice);

  Program h_program;
  h_program.add(LoadVector(a0,v0,BlockEltStart,VecWidth));
  h_program.add(LoadVector(a1,v1,BlockEltStart,VecWidth));
  h_program.add(Add(v0,v1,v2));
  h_program.add(StoreVector(a2, v2, BlockEltStart,VecWidth));

  printf("%d %d\n", *((uint16_t*)&h_program._ops[0]), *((uint16_t*) &h_program._ops[2]));
  printf("program length: %d\n", h_program.size());
  printf("load size %d\n", sizeof(LoadVector));
  printf("store size %d\n", sizeof(StoreVector));
  printf("add size %d\n", sizeof(Add));

  //  for (int i = 1; i <= N; i *= 2) {
  int total_blocks = divup(N, kRegisterWidth);
  dim3 blocks;
  blocks.x = int(ceil(sqrt(total_blocks)));
  blocks.y = int(ceil(sqrt(total_blocks)));
  blocks.z = 1;

  dim3 threads;
  threads.x = kThreadsX;
  threads.y = kThreadsY;
  threads.z = 1;

  fprintf(stderr, "%d %d %d; %d %d %d\n", blocks.x, blocks.y, blocks.z, threads.x, threads.y,
          threads.z);
  double st = Now();
  run<<<blocks, threads>>>(h_program.to_gpu(),
  		           h_program.size(),
    		           d_values,
    		           n_values, 0, 0);
  hipDeviceSynchronize();
  CHECK_CUDA();
  double ed = Now();
  fprintf(stderr, "%d elements in %.5f seconds; %.5f GFLOPS\n", N, ed - st, N * 1e-9 / (ed - st));
//  }

  float* ad = a.get_host_data();
  printf("%f %f %f\n", ad[0], ad[10], ad[N - 200]);
  float* bd = b.get_host_data();
  printf("%f %f %f\n", bd[0], bd[10], bd[N - 200]);
  float* cd = c.get_host_data();
  for (int i = 0; i < min(1024, N); ++i) {
    printf("%.0f ", cd[i]);
    if (i % 64 == 63) {
      printf("\n");
    }
  }

  for (int i = 0; i < N; ++i) {
    if (cd[i] == 0) {
      printf("ZERO at %d\n", i);
      break;
    }
  }
  return 0;
}
